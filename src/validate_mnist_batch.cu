#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"
#include "npz_converter.h"

using namespace marian;
using namespace keywords;

int main(int argc, char** argv) {
  
  hipSetDevice(0);
  
  const size_t IMAGE_SIZE = 784;
  const size_t LABEL_SIZE = 10;
  const size_t BATCH_SIZE = 24;
  int numofdata;

  std::cerr << "Loading test set...";
  std::vector<float> testImages = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", numofdata, IMAGE_SIZE);
  std::vector<float> testLabels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte", numofdata, LABEL_SIZE);
  std::cerr << "\tDone." << std::endl;

  std::cerr << "Loading model params...";
  NpzConverter converter("../scripts/test_model_multi/model.npz");

  std::vector<float> wData1;
  Shape wShape1;
  converter.Load("weights1", wData1, wShape1);
  
  std::vector<float> bData1;
  Shape bShape1;
  converter.Load("bias1", bData1, bShape1);
  
  std::vector<float> wData2;
  Shape wShape2;
  converter.Load("weights2", wData2, wShape2);
  
  std::vector<float> bData2;
  Shape bShape2;
  converter.Load("bias2", bData2, bShape2);

  auto initW1 = [wData1](Tensor t) {
    t.set(wData1);
  };

  auto initB1 = [bData1](Tensor t) {
    t.set(bData1);
  };
  
  auto initW2 = [wData2](Tensor t) {
    t.set(wData2);
  };

  auto initB2 = [bData2](Tensor t) {
    t.set(bData2);
  };

  std::cerr << "\tDone." << std::endl;


  auto x = input(shape={whatevs, IMAGE_SIZE}, name="X");
  auto y = input(shape={whatevs, LABEL_SIZE}, name="Y");

  auto w1 = param(shape={IMAGE_SIZE, 100}, name="W0", init=initW1);
  auto b1 = param(shape={1, 100}, name="b0", init=initB1);
  auto w2 = param(shape={100, LABEL_SIZE}, name="W1", init=initW2);
  auto b2 = param(shape={1, LABEL_SIZE}, name="b1", init=initB2);

  std::cerr << "Building model...";
  auto layer1 = tanh(dot(x, w1) + b1);
  auto layer2 = softmax(dot(layer1, w2) + b2, axis=1, name="layer2");
  auto cost = -mean(sum(y * log(layer2), axis=1), axis=0);

  std::cerr << "Done." << std::endl;

  Tensor xt({BATCH_SIZE, IMAGE_SIZE});

  size_t acc = 0;
  size_t startId = 0;
  size_t endId = startId + BATCH_SIZE;

  while (endId < numofdata) {
    std::vector<float> tmp(testImages.begin() + (startId * IMAGE_SIZE),
                           testImages.begin() + (endId * IMAGE_SIZE));
    xt << tmp;
    x = xt;

    cost.forward(BATCH_SIZE);

    std::vector<float> results(LABEL_SIZE * BATCH_SIZE);
    results << layer2.val();

    for (size_t i = 0; i < BATCH_SIZE * LABEL_SIZE; i += LABEL_SIZE) {
      size_t correct = 0;
      size_t predicted = 0;
      for (size_t j = 0; j < LABEL_SIZE; ++j) {
        if (testLabels[startId * LABEL_SIZE + i + j]) correct = j;
        if (results[i + j] > results[i + predicted]) predicted = j;
      }
      /*std::cerr << "CORRECT: " << correct << " PREDICTED: " << predicted << std::endl;*/
      acc += (correct == predicted);
    }

    startId += BATCH_SIZE;
    endId += BATCH_SIZE;
  }
  if (endId != numofdata) {
    endId = numofdata;
    if (endId - startId > 0) {
      std::vector<float> tmp(testImages.begin() + (startId * IMAGE_SIZE),
                             testImages.begin() + (endId * IMAGE_SIZE));
      xt << tmp;
      x = xt;

      cost.forward(endId - startId);

      std::vector<float> results(LABEL_SIZE * BATCH_SIZE);
      results << layer2.val();

      for (size_t i = 0; i < (endId - startId) * LABEL_SIZE; i += LABEL_SIZE) {
        size_t correct = 0;
        size_t predicted = 0;
        for (size_t j = 0; j < LABEL_SIZE; ++j) {
          if (testLabels[startId * LABEL_SIZE + i + j]) correct = j;
          if (results[i + j] > results[i + predicted]) predicted = j;
        }
        acc += (correct == predicted);
      }
    }
  }
  std::cerr << "ACC: " << float(acc)/numofdata << std::endl;
  
  float eta = 0.1;
  for (size_t j = 0; j < 10; ++j) {
    for(size_t i = 0; i < 60; ++i) {    
      cost.backward();
    
      auto update_rule = _1 -= eta * _2;
      Element(update_rule, w1.val(), w1.grad());
      Element(update_rule, b1.val(), b1.grad());
      Element(update_rule, w2.val(), w2.grad());
      Element(update_rule, b2.val(), b2.grad());
      
      cost.forward(BATCH_SIZE);
    }
    std::cerr << "Epoch: " << j << std::endl;
    std::vector<float> results;
    results << layer2.val();
    
    size_t acc = 0;
    for (size_t i = 0; i < testLabels.size(); i += LABEL_SIZE) {
      size_t correct = 0;
      size_t proposed = 0;
      for (size_t j = 0; j < LABEL_SIZE; ++j) {
        if (testLabels[i+j]) correct = j;
        if (results[i + j] > results[i + proposed]) proposed = j;
      }
      acc += (correct == proposed);
    }
    std::cerr << "Cost: " << cost.val()[0] <<  " - Accuracy: " << float(acc) / BATCH_SIZE << std::endl;
  }


  return 0;
}
