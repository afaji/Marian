#include <algorithm>
#include <chrono>
#include <iomanip>
#include <cstdio>
#include <boost/timer/timer.hpp>

#include "marian.h"
#include "mnist.h"
#include "optimizers.h"

using namespace marian;
using namespace keywords;

const size_t IMAGE_SIZE = 784;
const size_t LABEL_SIZE = 10;
int BATCH_SIZE = 200;

ExpressionGraph build_graph(const std::vector<int>& dims) {
  std::cerr << "Building model... ";
  boost::timer::cpu_timer timer;
    
  ExpressionGraph g;
  auto x = named(g.input(shape={whatevs, IMAGE_SIZE}), "x");
  auto y = named(g.input(shape={whatevs, LABEL_SIZE}), "y");
  
  std::vector<Expr> layers, weights, biases;
  for(int i = 0; i < dims.size()-1; ++i) {
    int in = dims[i];
    int out = dims[i+1];
      
    if(i == 0) {
      layers.emplace_back(dropout(x, value=0.2));
    }
    else {
      layers.emplace_back(dropout(relu(dot(layers.back(), weights.back()) + biases.back()), value=0.5));
    }
    
    weights.emplace_back(
      named(g.param(shape={in, out}, init=uniform()), "W" + std::to_string(i)));
    biases.emplace_back(
      named(g.param(shape={1, out}, init=zeros), "b" + std::to_string(i)));
  }

  auto scores = named(dot(layers.back(), weights.back()) + biases.back(),
                      "scores");
  
  auto cost = mean(cross_entropy(scores, y), axis=0);
  auto costreg = named(
    cost, "cost"
  );

  // If we uncomment the line below, this will just horribly diverge.
  // auto dummy_probs = named(softmax(scores), "dummy_probs");

  std::cerr << timer.format(5, "%ws") << std::endl;
  return g;
}

void shuffle(std::vector<float>& x, std::vector<float>& y, size_t dimx, size_t dimy) {
  std::srand(std::time(0));
  std::vector<size_t> ind;
  for(size_t i = 0; i < x.size() / dimx; ++i) {
    ind.push_back(i);
  }
  
  std::random_shuffle(ind.begin(), ind.end());
  
  std::vector<float> xShuffled(x.size());
  std::vector<float> yShuffled(y.size());
  
  int j = 0;
  for(auto i : ind) {
    std::copy(x.begin() + j * dimx, x.begin() + j * dimx + dimx, xShuffled.begin() + i * dimx);
    std::copy(y.begin() + j * dimy, y.begin() + j * dimy + dimy, yShuffled.begin() + i * dimy);
    j++;
  }
  
  x = xShuffled;
  y = yShuffled;
  
}

float accuracy(const std::vector<float> pred, const std::vector<float> labels) {
  size_t acc = 0;
  for (size_t i = 0; i < labels.size(); i += LABEL_SIZE) {
    size_t correct = 0;
    size_t proposed = 0;
    for (size_t j = 0; j < LABEL_SIZE; ++j) {
      if (labels[i + j])
        correct = j;
      if (pred[i + j] > pred[i + proposed])
        proposed = j;
    }
    acc += (correct == proposed);
  }
  return float(acc) / (labels.size() / LABEL_SIZE);
}

int main(int argc, char** argv) {

  std::cerr << std::setprecision(4) << std::fixed;

  int trainRows, testRows;
  
  std::cerr << "Loading train set...";
  std::vector<float> trainImages = datasets::mnist::ReadImages("../examples/mnist/train-images-idx3-ubyte", trainRows, IMAGE_SIZE);
  std::vector<float> trainLabels = datasets::mnist::ReadLabels("../examples/mnist/train-labels-idx1-ubyte", trainRows, LABEL_SIZE);
  std::cerr << "Done." << std::endl;
  
  std::cerr << "Loading test set...";
  std::vector<float> testImages = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", testRows, IMAGE_SIZE);
  std::vector<float> testLabels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte", testRows, LABEL_SIZE);
  std::cerr << "Done." << std::endl;

  ExpressionGraph g = build_graph({IMAGE_SIZE, 2048, 2048, LABEL_SIZE});
  
  std::ofstream viz("mnist_benchmark.dot");
  viz << g.graphviz() << std::endl;
  viz.close();
  
  Tensor xt({BATCH_SIZE, IMAGE_SIZE});
  Tensor yt({BATCH_SIZE, LABEL_SIZE});
  
  boost::timer::cpu_timer total;
  Adam opt(0.0002);
  for(int i = 1; i <= 50; ++i) {
    boost::timer::cpu_timer timer;
    shuffle(trainImages, trainLabels, IMAGE_SIZE, LABEL_SIZE);
    float cost = 0;
    float acc = 0;
    for(int j = 0; j < trainRows / BATCH_SIZE; j++) {
      size_t xBatch = IMAGE_SIZE * BATCH_SIZE;
      auto xbegin = trainImages.begin() + j * xBatch;
      auto xend = xbegin + xBatch;
      xt.set(xbegin, xend);
      
      size_t yBatch = LABEL_SIZE * BATCH_SIZE;
      auto ybegin = trainLabels.begin() + j * yBatch;
      auto yend = ybegin + yBatch;
      std::vector<float> ytv(ybegin, yend);
      yt.set(ytv);
      
      g["x"] = xt;
      g["y"] = yt;
      
      opt(g, BATCH_SIZE);
        
      cost += (g["cost"].val()[0] * BATCH_SIZE) / trainRows;
      
      std::vector<float> bResults;
      bResults << g["scores"].val();
      
      acc += (accuracy(bResults, ytv) * BATCH_SIZE) / trainRows;
    }
    std::cerr << "Epoch: " << i << " - Cost: " << cost << " - Accuracy: " << acc << " - " << timer.format(3, "%ws") << std::endl;
  }
  std::cerr << "Total: " << total.format(3, "%ws") << std::endl;

  std::vector<float> results;
  for(int j = 0; j < testRows / BATCH_SIZE; j++) {
    size_t xBatch = IMAGE_SIZE * BATCH_SIZE;
    auto xbegin = testImages.begin() + j * xBatch;
    auto xend = xbegin + xBatch;
    xt.set(xbegin, xend);
    yt.set(0);  
      
    g["x"] = xt;
    g["y"] = yt;
    
    g.inference(BATCH_SIZE);
    
    std::vector<float> bResults;
    bResults << g["scores"].val();
    results.insert(results.end(), bResults.begin(), bResults.end());
  }
  
  std::cerr << "Accuracy: " << accuracy(results, testLabels) << std::endl;
  
  return 0;
}
