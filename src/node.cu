#include "hip/hip_runtime.h"
#include "node.h"

namespace marian {

// for backward_numeric
void Node::calc_numeric_grad(
		  Float delta,
		  Tensor input,
		  Tensor grad,
		  const std::vector<float> &prevCalcGrad
		  )
{
  using namespace std;

	  size_t inputSize = GetTotalSize(input.shape());
	  size_t valSize = GetTotalSize(val_.shape());

	  UTIL_THROW_IF2(inputSize != GetTotalSize(grad.shape()),
			  	  "inputSize != gradSize:" << inputSize << "!=" << GetTotalSize(grad.shape()));
	  UTIL_THROW_IF2(valSize != GetTotalSize(adj_.shape()),
			  	  "valSize != adjSize :" << valSize << "!=" << GetTotalSize(adj_.shape()));

	  cerr	<< "inputSize=grad=" << Debug(input.shape())<< "=" << inputSize << " "
			<< "valSize=adj_=" << Debug(val_.shape()) << "=" << valSize
			<< endl;

	  //cerr << "input=" << input.Debug() << endl;
	  //cerr << "adj_=" << adj_.Debug() << endl;

	  std::vector<float> origGrad(inputSize);
	  thrust::copy(grad.begin(), grad.end(), origGrad.begin());
	  cerr << "origGrad=" << grad.Debug() << endl;
	  //output("diffGrad", diffGrad);

	  //output("prevCalcGrad", prevCalcGrad.begin(), prevCalcGrad.end());

	  std::vector<float> inputVec(inputSize);
	  thrust::copy(input.begin(), input.end(), inputVec.begin());
	  //output("inputVec", inputVec);

	  std::vector<float> newVal(inputSize, 0);

	  // LOOP thru each element in input & add delta
	  for (size_t inputInd = 0; inputInd < inputSize; ++inputInd) {
		  inputVec[inputInd] += delta;
		  thrust::copy(inputVec.begin(), inputVec.end(), input.begin());
		  //output("input", input.begin(), input.end());

		  forward();

		  for (size_t i = 0; i < valSize; ++i) {
			  newVal[inputInd] += val_[i];
		  }
		  //output("val_", val_.begin(), val_.end());

		  inputVec[inputInd] -= delta;
	  }

	  // orig value
	  thrust::copy(inputVec.begin(), inputVec.end(), input.begin());
	  forward();

	  float sumValOrig = 0;
	  for (size_t i = 0; i < valSize; ++i) {
		  sumValOrig += val_[i];
	  }

	  //output("newVal", newVal.begin(), newVal.end());

	  // calc gradient
	  //cerr << "adj_=" << adj_.Debug() << endl;
	  std::vector<float> adjVec(valSize);
	  thrust::copy(adj_.begin(), adj_.end(), adjVec.begin());

	  std::vector<float> numericalGrad(inputSize);
	  for (size_t i = 0; i < numericalGrad.size(); ++i) {
		  numericalGrad[i] = (newVal[i] - sumValOrig) / delta;
	  }

	  broadcast(numericalGrad, adjVec);
	  //std::cerr << "broadcast size=" << numericalGrad.size() << " " << adjVec.size() << std::endl;
	  //output("adjVec=", adjVec.begin(), adjVec.end());

	  for (size_t i = 0; i < numericalGrad.size(); ++i) {
		  numericalGrad[i] *= adjVec[i];
		  numericalGrad[i] += prevCalcGrad[i];
	  }

	  //output("prevCalcGrad=", prevCalcGrad.begin(), prevCalcGrad.end());
	  //output("adjVec=", adjVec.begin(), adjVec.end());

	  // set grad results
	  thrust::copy(numericalGrad.begin(), numericalGrad.end(), grad.begin());
	  cerr << "numericalGrad=" << grad.Debug() << endl;
	  //output("numericalGrad", numericalGrad);

	  // print out diff between origGrad and numericalGrad
	  std::vector<float> diff(inputSize);
	  for (size_t i = 0; i < origGrad.size(); ++i) {
		  diff[i] = origGrad[i] - numericalGrad[i];
	  }
	  cerr << "L2-norm of difference=" << L2Norm(diff) << endl << endl;

	  // put back origGrad
	  thrust::copy(origGrad.begin(), origGrad.end(), grad.begin());
}

float Node::L2Norm(const std::vector<float> &vec) const
{
  float ret = 0;
  for (size_t i = 0; i < vec.size(); ++i) {
	  ret += vec[i] * vec[i];
  }
  return sqrt(ret);
}

std::vector<float> Node::StoreTensorInVec(Tensor tensor)
{
  size_t totSize = GetTotalSize(tensor.shape());
  std::vector<float> vec(totSize);
  thrust::copy(tensor.begin(), tensor.end(), vec.begin());
  return vec;
}

void Node::broadcast(const std::vector<float> &largeVec, std::vector<float> &smallVec)
{
	size_t largeSize = largeVec.size();
	size_t smallSize = smallVec.size();

    UTIL_THROW_IF2(largeSize < smallSize,
    		"largeSize < smallSize:" << largeSize << "<" << smallSize);
    UTIL_THROW_IF2(largeSize % smallSize,
    		"largeSize % smallSize != 0:" << largeSize << " " << smallSize);

    smallVec.resize(largeSize);
    for (size_t i = smallSize; i < largeSize; i += smallSize) {
    	std::copy(smallVec.begin(), smallVec.begin() + smallSize, smallVec.begin() + i);
    }
}

}

