#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hipDNN.h>

#include <boost/timer/timer.hpp>

#include "tensor.h"
#include "tensor_operators.h"
#include "param_initializers.h"

using namespace marian;

template <class F>
void testForward(F f, size_t l,
                 const Shape& shape,
                 const std::string& desc) {
    Tensor in(shape);
    Tensor out(shape);
    
    uniform(-5, 5)(in);

    std::cout << desc << ": " << std::flush;
    boost::timer::cpu_timer timer;
    for(int i = 0; i < l; ++i) {
      f(out, in);
      if(i % 100 == 0)
        std::cout << "." << std::flush;
    }
    std::cout << timer.format(5, "%ws") << std::endl;
}

template <class F>
void testBackward(F f, size_t l,
                  const Shape& shape,
                  const std::string& desc) {
    Tensor in(shape);
    Tensor adj(shape, 1);
    Tensor grad(shape);
    
    uniform(-5, 5)(in);

    std::cout << desc << ": " << std::flush;
    boost::timer::cpu_timer timer;
    for(int i = 0; i < l; ++i) {
      f(grad, adj, in);
      if(i % 100 == 0)
        std::cout << "." << std::flush;
    }
    std::cout << timer.format(5, "%ws") << std::endl;
}

int main() {
    int l = 1000;
    
    std::vector<Shape> shapes = {
        {1000, 1000},
        {80, 50000},
        {50000, 80},
    };
    
    for(auto& shape : shapes) {
        std::cout << "Testing shape: " << shape[0] << "x" << shape[1] << std::endl << std::endl; 
        
        std::cout << "Softmax forward" << std::endl;
        testForward(CudnnSoftmax, l, shape, "CuDNN ");
        testForward(Softmax, l, shape, "Marian");
        std::cout << std::endl;
        
        std::cout << "Softmax backward" << std::endl;
        testBackward(CudnnSoftmaxGrad, l, shape, "CuDNN ");
        testBackward(SoftmaxGrad, l, shape, "Marian");
        std::cout << std::endl;
        
        std::cout << "Log-softmax backward" << std::endl;
        testBackward(CudnnLogSoftmaxGrad, l, shape, "CuDNN ");
        testBackward(LogSoftmaxGrad, l, shape, "Marian");
        std::cout << std::endl;
    }
    return 0;
}