#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include "marian.h"
#include "expression_graph.h"
#include "keywords.h"
#include "definitions.h"


float Rand()
{
	float LO = -10;
	float HI = +20;
	float r3 = LO + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(HI-LO)));
	return r3;
}

int main(int argc, char** argv)
{
  using namespace std;
  using namespace marian;
  using namespace keywords;

  int input_size = 10;
  int output_size = 10;
  int batch_size = 25;

  // define graph
  ExpressionGraph g;
  Expr inExpr = g.input(shape={batch_size, input_size});
  Expr labelExpr = g.input(shape={batch_size, output_size});

  Expr inExpr2 = g.input(shape={batch_size, input_size});
  Expr inExpr3 = g.input(shape={input_size, batch_size});

  vector<Expr> expr;

  expr.emplace_back(inExpr + inExpr2);
  expr.emplace_back(inExpr - expr.back());
  expr.emplace_back(inExpr * expr.back());
  expr.emplace_back(inExpr / expr.back());
  expr.emplace_back(reluplus(inExpr, expr.back()));

  //expr.emplace_back(dot(inExpr, inExpr3));

  expr.emplace_back(tanh(expr.back()));
  expr.emplace_back(-expr.back());
  expr.emplace_back(logit(expr.back()));
  expr.emplace_back(relu(expr.back()));
  expr.emplace_back(log(expr.back()));
  expr.emplace_back(exp(expr.back()));
  expr.emplace_back(softmax(expr.back()));

  Expr ceExpr = cross_entropy(expr.back(), labelExpr);
  Expr cost = mean(ceExpr, axis=0);

  // create data
  //srand(0);
  srand(time(NULL));
  std::vector<float> values(batch_size * input_size);
  generate(begin(values), end(values), Rand);

  std::vector<float> labels(batch_size * input_size);
  generate(begin(labels), end(labels), Rand);

  Tensor inTensor({batch_size, input_size});
  thrust::copy(values.begin(), values.end(), inTensor.begin());

  Tensor labelTensor({batch_size, input_size});
  thrust::copy(labels.begin(), labels.end(), labelTensor.begin());

  inExpr = inTensor;
  labelExpr = labelTensor;

  // for binary expressions
  std::vector<float> values2(batch_size * input_size);
  generate(begin(values2), end(values2), Rand);
  Tensor inTensor2({batch_size, input_size});
  thrust::copy(values2.begin(), values2.end(), inTensor2.begin());

  inExpr2 = inTensor2;

  Tensor inTensor3({input_size, batch_size});
  thrust::copy(values2.begin(), values2.end(), inTensor3.begin());

  inExpr3 = inTensor3;

  // train
  g.forward(batch_size);
  //g.backward();
  g.backward_debug(0.001);

  std::cout << g.graphviz() << std::endl;

  /*
  std::cerr << "inTensor=" << inTensor.Debug() << std::endl;

  Tensor outTensor = outExpr.val();
  std::cerr << "outTensor=" << outTensor.Debug() << std::endl;

  Tensor outGrad = outExpr.grad();
  std::cerr << "outGrad=" << outGrad.Debug() << std::endl;
  */

}
