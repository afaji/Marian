#include "hip/hip_runtime.h"
// This file is part of the Marian toolkit.
// Marian is copyright (c) 2016 Marcin Junczys-Dowmunt.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "expression_operators.h"
#include "node_operators.h"

namespace marian {

Expr named(Expr a, const std::string& name) {
  a.node()->set_name(name);
  a.graph()->add_named_node(a, name);
  return a;
}

Expr logit(Expr a) {
  return Expr(a.graph(), new LogitNodeOp(a));
}

Expr tanh(Expr a) {
  return Expr(a.graph(), new TanhNodeOp(a));
}

Expr relu(Expr a) {
  return Expr(a.graph(), new ReLUNodeOp(a));
}

Expr log(Expr a) {
  return Expr(a.graph(), new LogNodeOp(a));
};

Expr exp(Expr a) {
  return Expr(a.graph(), new ExpNodeOp(a));
};

Expr operator-(Expr a) {
  return Expr(a.graph(), new NegNodeOp(a));
};

Expr softmax(Expr a) {
  return Expr(a.graph(), new SoftmaxNodeOp(a));
}

Expr logsoftmax(Expr a) {
  return Expr(a.graph(), new LogSoftmaxNodeOp(a));
}

Expr argmax(Expr a) {
  return Expr(a.graph(), new ArgmaxNodeOp(a));
}

/*********************************************************/

Expr operator+(Expr a, Expr b) {
  return Expr(a.graph(), new PlusNodeOp(a, b));
}

Expr operator-(Expr a, Expr b) {
  return Expr(a.graph(), new MinusNodeOp(a, b));
}

Expr operator*(Expr a, Expr b) {
  return Expr(a.graph(), new MultNodeOp(a, b));
}

Expr operator/(Expr a, Expr b) {
  return Expr(a.graph(), new DivNodeOp(a, b));
}

Expr dot(Expr a, Expr b) {
  return Expr(a.graph(), new DotNodeOp(a, b));
}

Expr reluplus(Expr a, Expr b) {
  return Expr(a.graph(), new ReLUPlusNodeOp(a, b));
}

Expr cross_entropy(Expr a, Expr b) {
  return Expr(a.graph(), new CrossEntropyNodeOp(a, b));
}

}
