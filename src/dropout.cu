#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "dropout.h"

namespace marian {

__global__ void gInitCurandStates(hiprandState* states, unsigned int seed) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &states[tid]);
}

unsigned Bernoulli::seed = time(0);

}